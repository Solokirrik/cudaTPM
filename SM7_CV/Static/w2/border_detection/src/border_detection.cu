/*
 ============================================================================
 Name        : border_detection.cu
 Author      : Solo
 Version     :
 Copyright   : 
 Description : CUDA compute reciprocals
 ============================================================================
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <numeric>
#include <stdlib.h>
#include <string>
#include <fstream>
#include <sstream>
#include <iterator>
#include <math.h>

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

static const int K[25] = {2, 4, 5, 4, 2,
					  4, 9, 12, 9, 4,
					  5, 12, 15, 12, 5,
					  4, 9, 12, 9, 4,
					  2, 4, 5, 4, 2};
static const int Gx[9] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
static const int Gy[9] = {-1, -2, -1, 0, 0, 0, 1, 2, 1};
/**
 * CUDA kernel that computes reciprocal values for a given vector
 */
__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
	unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < vectorSize)
		data[idx] = 1.0/data[idx];
}

__global__ void smoothingKernel(int *data, float *outdata, int *K, unsigned long vectorSize, unsigned k_w, unsigned k_h) {
	unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < vectorSize){
		float sigma_m = 1.0/159;
		int up = -2;
		int bot = 3;
		int left = -2;
		int right = 3;

		int mtrx_w = 5;
		int new_cen = mtrx_w / 2;

		int j = idx % k_w;
		int i = idx / k_w;

		if (i < -up){
			up = -i;
		}
		else if (i >  k_h - bot){
			bot = k_h - i;
		}
		if(j < -left){
			left = -j;
		}
		else if (j > k_w - right){
			right = k_w - j;
		}
		for(int k = up; k < bot; k++){
			for(int l = left; l < right; l++){
				outdata[idx] += sigma_m * K[(k + new_cen)*mtrx_w + l + new_cen] * data[(i + k)*k_w + (j + l)];
			}
		}
		outdata[idx] = int(outdata[idx]);
	}
}

__global__ void gradientKernel(float *data, float *outdata, int *Gx, int *Gy, unsigned long vectorSize, unsigned k_w, unsigned k_h) {
	unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < vectorSize){
		int up = -1;
		int bot = 2;
		int left = -1;
		int right = 2;

		int mtrx_w = 3;
		int new_cen = mtrx_w / 2;
		int j = idx % k_w;
		int i = idx / k_w;
		float Gxx = 0;
		float Gyy = 0;

		if (i < -up){
			up = -i;
		}
		else if (i >  k_h - bot){
			bot = k_h - i;
		}
		if(j < -left){
			left = -j;
		}
		else if (j > k_w - right){
			right = k_w - j;
		}

		for(int k = up; k < bot; k++){
			for(int l = left; l < right; l++){
				Gxx += Gx[(k + new_cen) * mtrx_w + l + new_cen] * data[(i + k)*k_w + (j + l)];
				Gyy += Gy[(k + new_cen) * mtrx_w + l + new_cen] * data[(i + k)*k_w + (j + l)];
				outdata[idx] = int(sqrt(pow(Gxx, 2) + pow(Gyy, 2)));
			}
		}
		outdata[idx] = int(outdata[idx]);
	}
}

__global__ void minmaxfilterKernel(float *data, int *outdata, unsigned long vectorSize) {
	unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < vectorSize){
		if(data[idx] < 0){
			outdata[idx] = 0;
		}
		else if(data[idx] > 255){
			outdata[idx] = 255;
		}
		outdata[idx] = int(data[idx]);
	}
}

/**
 * Host function that copies the data and launches the work on GPU
 */
int *gpuReciprocal(int *data, unsigned long size, unsigned width, unsigned height)
{
	int *rc = new int[size];
	int *gpuData;
	float *gpuOutData1, *gpuOutData2;
	int *gpuK, *gpuGx, *gpuGy;

	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuK, sizeof(K)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuGx, sizeof(Gx)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuGy, sizeof(Gy)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuData, sizeof(int)*size));
	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuOutData1, sizeof(float)*size));
	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuOutData2, sizeof(float)*size));

	CUDA_CHECK_RETURN(hipMemcpy(gpuK, Gx, sizeof(K), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(gpuGx, Gx, sizeof(Gx), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(gpuGy, Gy, sizeof(Gy), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(gpuData, data, sizeof(int)*size, hipMemcpyHostToDevice));
	
	static const int BLOCK_SIZE = 256;
	const int blockCount = (size+BLOCK_SIZE-1) / BLOCK_SIZE;

	smoothingKernel<<<blockCount, BLOCK_SIZE>>>(gpuData, gpuOutData1, gpuK, size, width, height);
	gradientKernel<<<blockCount, BLOCK_SIZE>>>(gpuOutData1, gpuOutData2, gpuGx, gpuGy, size, width, height);
	minmaxfilterKernel<<<blockCount, BLOCK_SIZE>>>(gpuOutData2, gpuData, size);

	CUDA_CHECK_RETURN(hipMemcpy(rc, gpuData, sizeof(float)*size, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree(gpuGx));
	CUDA_CHECK_RETURN(hipFree(gpuGy));
	CUDA_CHECK_RETURN(hipFree(gpuData));
	CUDA_CHECK_RETURN(hipFree(gpuOutData1));
	CUDA_CHECK_RETURN(hipFree(gpuOutData2));
	return rc;
}

int main(void)
{
	std::string line;
	unsigned H = 0;
	unsigned W = 0;
	int *data;
	unsigned long WORK_SIZE = 0;
	
	std::ifstream myFile("Labyrinth_1.txt");
	if (myFile.is_open()) {
		H = std::count(std::istreambuf_iterator<char>(myFile), std::istreambuf_iterator<char>(), '\n');
		myFile.seekg(0, myFile.beg);
		//W = std::count(std::istreambuf_iterator<char>(myFile), std::istreambuf_iterator<char>(), ' ') / H + 1; // 425ms

		getline(myFile, line);
		std::stringstream stream(line);	// 13ms
		int n;
		while (stream >> n) {
			W++;
		}
		std::cout << "lines:\t" << H << std::endl;
		std::cout << "values:\t" << W << std::endl;

		WORK_SIZE = W*H;
		data = new int[WORK_SIZE];

		myFile.seekg(0, myFile.beg);
		int i = 0;
		while (i < WORK_SIZE) {
			myFile >> data[i];
			i++;
		}
		std::cout << "Reading done" << std::endl;
	}
	else {
		std::cout << "File not found" << std::endl;
	}

	int *recGpu = gpuReciprocal(data, WORK_SIZE, W, H);

	for (int i = 0; i < H; i++) {
		for (unsigned j = 0; j < W; ++j) {
			std::cout << recGpu[i * W + j] << " ";
		}
		std::cout << std::endl;
	}

	/* Free memory */
	delete[] data;
	delete[] recGpu;

	return 0;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}
