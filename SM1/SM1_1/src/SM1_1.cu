/*
 ============================================================================
 Name        : SM1.cu
 Author      : Solo
 Version     :
 Copyright   : 
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <hip/hip_runtime.h>
#include <numeric>
#include <stdlib.h>
#include <iostream>
#include <fstream>
//#include <string>
#include <iomanip>

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

static const int time_s = 10;
static const float dx = 0.1;
static const float dx2 = 0.01;
static const float dt = 0.001;
static const float st_len = 100.0;

__global__ void reciprocalKernel(float *k_cur, float *k_next, unsigned vectorSize) {
	unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx == 0) {
		k_next[idx] = k_cur[idx] + 5;
	} else if (idx < vectorSize - 1) {
		k_next[idx] = (k_cur[idx + 1] - 2 * k_cur[idx] + k_cur[idx - 1]) * dt / dx2 + k_cur[idx];
	}
}


float *gpuReciprocal(float *data, unsigned size)
{
	float ht_count = time_s / dt;
	float GPUTime = 0.0f;

	float *buf;
	float *GPUcur;
	float *GPUnext;
	float *rc = new float[size];

	hipEvent_t start, stop;
	size_t mem_size = sizeof(float) * size;

	CUDA_CHECK_RETURN(hipMalloc((void **)&GPUcur, mem_size));
	CUDA_CHECK_RETURN(hipMalloc((void **)&GPUnext, mem_size));
	CUDA_CHECK_RETURN(hipMemcpy(GPUcur, data, mem_size, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(GPUnext, data, mem_size, hipMemcpyHostToDevice));

	static const int BLOCK_SIZE = 256;
	const int blockCount = (size+BLOCK_SIZE-1)/BLOCK_SIZE;

	hipEventCreate (&start);
	hipEventCreate (&stop);
	hipEventRecord (start , 0);

	for (int i = 0; i < ht_count; i++)
	{
		reciprocalKernel<<<blockCount, BLOCK_SIZE>>> (GPUcur, GPUnext, size);
		buf = GPUcur;
		GPUcur = GPUnext;
		GPUnext = buf;
	}

	hipEventRecord ( stop , 0);
	hipEventSynchronize ( stop );
	hipEventElapsedTime ( &GPUTime, start, stop);
	printf("GPU time: %.3f mS\n", GPUTime);

	CUDA_CHECK_RETURN(hipMemcpy(rc, GPUnext, sizeof(float)*size, hipMemcpyDeviceToHost));

	CUDA_CHECK_RETURN(hipFree(GPUcur));
	CUDA_CHECK_RETURN(hipFree(GPUnext));

	return rc;
}


void initData(float *data, unsigned size)
{
	for (unsigned i = 0; i < size; ++i)
		data[i] = 0.0;
}

int main(void)
{
	static const int WORK_SIZE = st_len / dx;
	float *data = new float[WORK_SIZE];

	initData (data, WORK_SIZE);

	float *recGpu = gpuReciprocal(data, WORK_SIZE);

	for (int i = 0; i < WORK_SIZE; i++){
		std::cout << std::setiosflags(std::ios::fixed) << std::setprecision(4) << std::setw(15)
			<< recGpu[i];
	}

	delete[] data;
	delete[] recGpu;

	return 0;
}

static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}
