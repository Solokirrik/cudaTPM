/*
 ============================================================================
 Name        : SM1_2.cu
 Author      : Solo
 Version     :
 Copyright   : 
 Description : CUDA compute reciprocals
 ============================================================================
 */


#include <hip/hip_runtime.h>
#include <numeric>
#include <stdlib.h>
#include <iostream>
#include <iomanip>
#include <math.h>

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

static const float a = 0.5;
static const float hx2 = 0.1;
static const float hy2 = 0.1;
static const float ht = 0.1;
static const float ht2 = 0.01;
static const float proc_time = 25;

static const int mem_ln = 99;
static const int mem_wd = 99;

__global__ void reciprocalKernel(float *k_prev, float *k_cur, float *k_next, float *k_pressure, unsigned k_LEN, unsigned k_WD) {
	unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;

	if ((idx % k_LEN != 0) && ((idx + 1) % k_LEN != 0) && (idx > k_LEN) && (idx < k_LEN * (k_WD - 1))) {
		k_next[idx] = 2 * k_cur[idx] - k_prev[idx]
			        + (a * ((k_cur[idx + 1] - 2 * k_cur[idx] + k_cur[idx - 1]) / hx2
			        + (k_cur[idx + k_LEN] - 2 * k_cur[idx] + k_cur[idx - k_LEN]) / hy2)
			        + k_pressure[idx]) * ht2;
	}
}

float *gpuReciprocal(float *hostData, float *hostPress, unsigned LENGHT, unsigned WIDTH)
{
	int size = LENGHT * WIDTH;
	float *rc;
	rc = new float[size];

	size_t data_size = sizeof(float) * size;
	hipEvent_t start, stop;

	float *buf;
	float *devPress;
	float *devPrev;
	float *devCur;
	float *devNext;

	float GPUTime = 0.0f;

	CUDA_CHECK_RETURN(hipMalloc((void **)&devPrev, data_size));
	CUDA_CHECK_RETURN(hipMalloc((void **)&devCur, data_size));
	CUDA_CHECK_RETURN(hipMalloc((void **)&devNext, data_size));
	CUDA_CHECK_RETURN(hipMalloc((void **)&devPress, data_size));

	CUDA_CHECK_RETURN(hipMemcpy(devPrev, hostData, data_size, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(devCur, hostData, data_size, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(devNext, hostData, data_size, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(devPress, hostPress, data_size, hipMemcpyHostToDevice));

	static const int BLOCK_SIZE = 256;
	const int blockCount = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

	hipEventCreate (&start );
	hipEventCreate (&stop );
	hipEventRecord (start , 0);

	for (int i = 0; i < proc_time/ht; i++){
		reciprocalKernel<<<blockCount, BLOCK_SIZE>>> (devPrev, devCur, devNext, devPress, LENGHT, WIDTH);
		buf = devPrev;
		devPrev = devCur;
		devCur = devNext;
		devNext = buf;
	}

	hipEventRecord ( stop , 0);
	hipEventSynchronize ( stop );
	hipEventElapsedTime ( &GPUTime, start, stop);
//	std::cout << std::setprecision(3) << "GPU time: " << GPUTime << " mS"<< std::endl;

	CUDA_CHECK_RETURN(hipMemcpy(rc, devCur, sizeof(float)*size, hipMemcpyDeviceToHost));

	CUDA_CHECK_RETURN(hipFree(devPrev));
	CUDA_CHECK_RETURN(hipFree(devCur));
	CUDA_CHECK_RETURN(hipFree(devNext));
	CUDA_CHECK_RETURN(hipFree(devPress));

	return rc;
}

// Gaussian distribution
void initGausPres(float *press, unsigned LENGHT, unsigned WIDTH)
{
	double r;
	double sigma = 5.0;
	double s = 2.0 * pow(sigma, 2);
	double sum = 0.0;

	int len2 = LENGHT / 2;
	int wid2 = WIDTH / 2;

	for (int x = -len2; x < len2 + 1; x++) {
		for (int y = -wid2; y < wid2 + 1; y++) {
			r = sqrt(x * x + y * y);
			press[(x + len2) * LENGHT + (y + wid2)] = exp(-pow(r, 2) / s) / (M_PI * s);
			sum += press[(x + len2) * LENGHT + (y + wid2)];
		}
	}
	// normalization
//	for (int i = 0; i < LENGHT; ++i)
//		for (int j = 0; j < WIDTH; ++j)
//			press[i * LENGHT + j] /= sum;

	// gain
	for (int i = 0; i < LENGHT; ++i)
		for (int j = 0; j < WIDTH; ++j)
			press[i * LENGHT + j] *= 1500;
}

void initData(float *data, unsigned size)
{
	for (unsigned i = 0; i < size; ++i)
		data[i] = 0;
}

int main(void)
{
	float *inpData;
	float *pressure;
	inpData = new float[mem_ln * mem_wd];
	pressure = new float[mem_ln * mem_wd];

	initData(inpData, mem_ln * mem_wd);
	initData(pressure, mem_ln * mem_wd);
	pressure[(mem_ln / 2) * mem_wd + mem_wd / 2] = 10;
//	initGausPres(pressure, mem_ln, mem_wd);

	float *recGpu = gpuReciprocal(inpData, pressure, mem_ln, mem_wd);

//	console output
//	std::cout << "External pressure" << std::endl;
//	for (int i = 0; i < mem_ln; i++) {
//		for (unsigned j = 0; j < mem_wd; ++j) {
//			std::cout << std::setiosflags(std::ios::fixed) << std::setprecision(3) << std::setw(10)
//				<< pressure[i * mem_ln + j];
//		}
//		std::cout << std::endl;
//	}
//	std::cout << "Result" << std::endl;
	for (unsigned i = 0; i < mem_ln; i++) {
		for (unsigned j = 0; j < mem_wd; ++j) {
			std::cout << std::setiosflags(std::ios::fixed) << std::setprecision(3) << std::setw(10)
				<< recGpu[i * mem_ln + j];
		}
		std::cout << std::endl;
	}

	delete[] inpData;
	delete[] recGpu;

	return 0;
}


static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}
