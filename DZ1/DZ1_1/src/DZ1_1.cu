/*
 ============================================================================
 Name        : DZ1_1.cu
 Author      : Solo
 Version     :
 Copyright   : 
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <numeric>
#include <stdlib.h>
#include <iomanip>

using namespace std;

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

static const int dT = 5;
static const unsigned tot_time = 100;
static const float len = 10.0;
static const float ht = 0.001;
static const float hx = 0.1;

static const float A = -ht/(hx*hx);
static const float B = 2*ht/(hx*hx) + 1;
static const float C = -ht/(hx*hx);

ofstream fout("temp.txt");

// ядро разогрева одного конца
__global__ void kernel_data_heat(float *data, float *newdata, unsigned vectorSize) {
	unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < vectorSize - 1){
		newdata[idx] = data[idx];
		if (idx == 0){
			newdata[idx] = data[idx] + dT * 0.1;
		}
	}
}

// ядро расчёта приближения
__global__ void kernel_data_calc(float *newData, float *stbData, float *bufData, unsigned vectorSize) {
	unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
	if ((idx > 0)&&(idx < vectorSize - 1)){
		newData[idx] = bufData[idx] + (stbData[idx] - (A*bufData[idx+1] + B*bufData[idx] + C*bufData[idx-1]));
	}
	else if(idx == vectorSize - 1){
		newData[idx] = 0;
	}
}

// вывод в файл
void print_to_file(float *recGpu, unsigned long a_width)
{
	for (unsigned long j = 0; j < a_width; ++j) {
		fout << recGpu[j] << " ";
	}
	fout << endl;
}

float *gpuReciprocal(float *hostData, unsigned size)
{
	float GPUTime = 0.0f;
	float *rc = new float[size];

	float *buf;			// буфер обмена обновления и нового приближения
	float *heatedData;	// буфер разогрева конца

	float *devStbData;	// установившиеся значение
	float *devNewData;	// обновлённые значения на шаге приближения
	float *devBufData;	// буфер приближения к базе

	hipEvent_t start, stop;
	size_t mem_size = sizeof(float) * size;

	CUDA_CHECK_RETURN(hipMalloc((void **)&devStbData, mem_size));
	CUDA_CHECK_RETURN(hipMalloc((void **)&devNewData, mem_size));
	CUDA_CHECK_RETURN(hipMalloc((void **)&devBufData, mem_size));
	CUDA_CHECK_RETURN(hipMalloc((void **)&buf, mem_size));
	CUDA_CHECK_RETURN(hipMalloc((void **)&heatedData, mem_size));

	CUDA_CHECK_RETURN(hipMemcpy(devStbData, hostData, mem_size, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(devNewData, hostData, mem_size, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(devBufData, hostData, mem_size, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(heatedData, hostData, mem_size, hipMemcpyHostToDevice));

	static const int BLOCK_SIZE = 256;
	const int blockCount = (size+BLOCK_SIZE-1)/BLOCK_SIZE;

	hipEventCreate (&start);
	hipEventCreate (&stop);
	hipEventRecord (start, 0);

	for (unsigned long i = 0; i < tot_time; i++) {
		hipMemcpy(heatedData, devStbData, mem_size, hipMemcpyDeviceToDevice);
		kernel_data_heat<<<blockCount, BLOCK_SIZE>>>(devStbData, heatedData, size);
//		обновление базовых значений
		hipMemcpy(devStbData, heatedData, mem_size, hipMemcpyDeviceToDevice);
		hipMemcpy(devBufData, devStbData, mem_size, hipMemcpyDeviceToDevice);
		hipMemcpy(devNewData, devStbData, mem_size, hipMemcpyDeviceToDevice);
//		прогон разогрева от нового базового значения
		for (unsigned long j = 0; j < 1000; j++) {
			kernel_data_calc<<<blockCount, BLOCK_SIZE>>>(devNewData, devStbData, devBufData, size);
//			обновление промежуточных значений температуры
			hipMemcpy(buf, devBufData, mem_size, hipMemcpyDeviceToDevice);
			hipMemcpy(devBufData, devNewData, mem_size, hipMemcpyDeviceToDevice);
			hipMemcpy(devNewData, buf, mem_size, hipMemcpyDeviceToDevice);
		}
//		запись нового базового значения
		hipMemcpy(devStbData, devNewData, mem_size, hipMemcpyDeviceToDevice);
		hipMemcpy(rc, devStbData, mem_size, hipMemcpyDeviceToHost);
		print_to_file(rc, size);
	}

	hipEventRecord ( stop , 0);
	hipEventSynchronize ( stop );
	hipEventElapsedTime ( &GPUTime, start, stop);
	cout << "\n" << GPUTime << endl;

	CUDA_CHECK_RETURN(hipMemcpy(rc, devStbData, mem_size, hipMemcpyDeviceToHost));

	CUDA_CHECK_RETURN(hipFree(devBufData));
	CUDA_CHECK_RETURN(hipFree(devNewData));
	CUDA_CHECK_RETURN(hipFree(devStbData));

	return rc;
}


void initData(float *data, unsigned width)
{
	for (unsigned i = 0; i < width; i++){
		data[i] = 0.0;
	}
	data[0] = 1;
	data[width - 1] = 0;
}

int main(void)
{
	const unsigned a_width = len / hx;
	float *data = new float[a_width];
	float *recGpu;
	initData(data, a_width);

	cout << "Init done" << endl;
	if(fout.is_open()){
		cout << "Writing data" << endl;

		recGpu = gpuReciprocal(data, a_width);

		cout << "\n" << "Data saved" << endl;
	} else {
		cout << "File could not be opened." << endl;
	}

	delete[] data;
	delete[] recGpu;

	return 0;
}


// Check the return value of the CUDA runtime API call and exit the application if the call has failed.
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}
