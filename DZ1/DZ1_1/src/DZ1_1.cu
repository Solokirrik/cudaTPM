/*
 ============================================================================
 Name        : DZ1_1.cu
 Author      : Solo
 Version     :
 Copyright   : 
 Description : CUDA compute reciprocals
 ============================================================================
 */


#include <hip/hip_runtime.h>
#include <numeric>
#include <stdlib.h>
#include <iostream>
#include <iomanip>

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

static const int time_s = 10;
static const float eps = 1e-3;
static const float st_len = 100.0;
static const float ht = 0.001;
static const float hx = 0.1;
static const float hx2 = 0.01;
static const float hthx2 = ht/hx2;
//static const float hx2ht = 0.5*hx2/ht;


__global__ void reciprocalKernel(float *k_cur, float *k_next, bool fl, unsigned k_size) {
	unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
	if ((idx == 0) && fl){
		k_cur[idx] = k_next[idx] + 5 * ht;
	} else	if (idx < k_size - 1) {
//		k_next[idx] = (k_cur[idx] + hthx2 * k_next[idx + 1] + hthx2 * k_next[idx - 1]) * hx2ht;
		k_cur[idx] = - hthx2 * k_next[idx + 1] + 2*hthx2*k_next[idx] + hthx2 * k_next[idx - 1];
	}
}


float *gpuReciprocal(float *hostData, unsigned size)
{
	float ht_count = time_s / ht;
	float GPUTime = 0.0f;
	float *rc = new float[size];
	float *devNext;
	float *devCur;
	float *buf;

	bool fl = false;

	hipEvent_t start, stop;
	size_t mem_size = sizeof(float) * size;

	CUDA_CHECK_RETURN(hipMalloc((void **)&devNext, mem_size));
	CUDA_CHECK_RETURN(hipMalloc((void **)&devCur, mem_size));
	CUDA_CHECK_RETURN(hipMemcpy(devNext, hostData, mem_size, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(devCur, hostData, mem_size, hipMemcpyHostToDevice));

	static const int BLOCK_SIZE = 256;
	const int blockCount = (size+BLOCK_SIZE-1)/BLOCK_SIZE;

	hipEventCreate (&start);
	hipEventCreate (&stop);
	hipEventRecord (start, 0);

	for (int i = 0; i < ht_count; i++) {
		for (int j = 0; j < 10000; j++) {
			reciprocalKernel<<<blockCount, BLOCK_SIZE>>>(devCur, devNext, fl,size);
			buf = devCur;
			devCur = devNext;
			devNext = buf;
		}
	}

	hipEventRecord ( stop , 0);
	hipEventSynchronize ( stop );
	hipEventElapsedTime ( &GPUTime, start, stop);
//	std::cout << std::setprecision(3) << "GPU time: " << GPUTime << " mS"<< std::endl;

	CUDA_CHECK_RETURN(hipMemcpy(rc, devNext, sizeof(float)*size, hipMemcpyDeviceToHost));

	CUDA_CHECK_RETURN(hipFree(devNext));
	CUDA_CHECK_RETURN(hipFree(devCur));

	return rc;
}


void initData(float *data, unsigned size)
{
	for (unsigned i = 0; i < size; ++i)
		data[i] = 0.0;
}

int main(void)
{
	static const int WORK_SIZE = st_len / hx;
	float *data = new float[WORK_SIZE];

	initData (data, WORK_SIZE);

	float *recGpu = gpuReciprocal(data, WORK_SIZE);

	for (int i = 0; i < WORK_SIZE; i++){
		std::cout << std::setiosflags(std::ios::fixed) << std::setprecision(4) << std::setw(15)
			<< recGpu[i];
	}
	std::cout << std::endl;

	delete[] data;
	delete[] recGpu;

	return 0;
}


// Check the return value of the CUDA runtime API call and exit the application if the call has failed.
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}
